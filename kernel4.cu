#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <pthread.h>
#include <unistd.h>

const int num_submatrix = 2;
const int numStreams = 2;
const int num_threads = numStreams;


struct thread_args{
	int threadId;
	unsigned long long overflowA;
	unsigned long long numSubMatrixA;
	unsigned long long subRows;
	unsigned long long subCols;
	unsigned long long m;
	unsigned long long n;
	unsigned long long k;
	unsigned long long y;
	unsigned long long i;
	float *C;
	float *A;

};

volatile int running_threads = 0;
pthread_mutex_t running_mutex = PTHREAD_MUTEX_INITIALIZER;


hipStream_t streams[numStreams];
float* b = 0;
float* a[num_threads];
float* c[num_threads];
float* a_h[num_threads];
float* c_h[num_threads];
struct thread_args targs[num_threads];
pthread_t threads[num_threads];
char threads_active[num_threads];
hipblasHandle_t handles[num_threads];



float * doMultiply2Matrices(
        int a1Rows, int a1Cols,  float * A1,
        int a2Rows, int a2Cols,  float * A2,
	float* C, hipStream_t cudaStream, hipblasHandle_t handle)
{

    float alpha = 1.0;
    float beta =  0.0;

    hipblasSetStream(handle, cudaStream) ;

    hipblasStatus_t stat = hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N,
                  a2Cols, a1Rows, a1Cols,
                  &alpha,
                  A2, a2Cols,
                  A1, a1Cols,
                  &beta,
                  C, a2Cols );
    printf("cublas status = %d\n", stat);

    return C ;


}

void PrintMatrix(char name[], int rows, int cols, const float* m){
  printf("%s\n", name);
  for(int row = 0; row < rows; ++row){
	for(int col = 0; col < cols; ++col){
		printf("%f ", m[row * cols + col]);
	}
	printf("\n");
  }
}


void copyElements(float* out, float* entry, unsigned long long eRows, unsigned long long eCols, unsigned long long oRows, unsigned long long oCols, unsigned long long x, unsigned long long y){
	for(unsigned long long i = 0; i < eRows; ++i){
		for(unsigned long long j = 0; j < eCols; ++j){
			out[x*eRows*oCols + (i*oCols) + (y*eCols + j)] = entry[i*eCols + j];
		}

	}



}


void msplitm(char transa, char transb, unsigned long long m, unsigned long long n, unsigned long long k, float alpha, float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    
    printf("entering msplitm \n");
    float* A_d;
    float* B_d;
    float* C_d;
    unsigned long long A_sz = m * k;
    unsigned long long B_sz = n * k;
    unsigned long long C_sz = m * n;
    unsigned long long MAX =  (unsigned long long )m* (unsigned long long) n / num_submatrix;
    
	MAX -= MAX % k;
	printf("MAX: %d\n", MAX);
	printf("B_sz: %d\n",B_sz);
	unsigned long long numSubMatrixB = B_sz / MAX;
	printf("SubmatriciesB: %d\n", numSubMatrixB);
	unsigned long long SMB_sz = B_sz / numSubMatrixB;
	printf("SMB_sz: %d\n", SMB_sz);
	unsigned long long subCols = B_sz / (numSubMatrixB * k);
	printf("subCols: %d\n", subCols);
	
	unsigned long long numSubMatrixA = A_sz / MAX;
	unsigned long long SMA_sz = A_sz / numSubMatrixA;
	unsigned long long subRows = A_sz / (numSubMatrixA * k);
	printf("subrows: %d\n", subRows);
	printf("SMA_sz: %d\n", SMA_sz);
	printf("submatriciesA: %d\n", numSubMatrixA);
	unsigned long long overflowA = m % subRows;
	unsigned long long overflowB = n % subCols;
	printf("overflowB: %d\n", overflowB);
	printf("overflowA: %d\n", overflowA);
	hipMalloc((void**) &b, sizeof(float) * subCols * k);
	for(int i = 0; i < numStreams; ++i){
		hipblasCreate(&handles[i]);
		hipStreamCreate(&streams[i]);
		hipMalloc((void**) &a[i], sizeof(float) * subRows * k);
		hipMalloc((void**) &c[i], sizeof(float) * subCols * subRows);
		hipHostMalloc((void**) &a_h[i], sizeof(float) * subRows * k, hipHostMallocDefault);
		hipHostMalloc((void**) &c_h[i], sizeof(float) * subCols * subRows, hipHostMallocDefault);
		threads_active[i] = 0;
	}

	float* temp3 = 0;
	
	hipHostMalloc((void**) &temp3, sizeof(float)*subCols * k , hipHostMallocDefault);
	for(unsigned long long i = 0; i < numSubMatrixB; ++i){
		int count = 0;
		if(overflowB == 0 && i == numSubMatrixB){
			break;
		}
	
		for(int j = 0; j < k; ++j){
			for(int x = 0; x < subCols; ++x){
				if(i * subCols + x < n){
					temp3[j * subCols + x] = B[j * n + (i*subCols + x)];
				}else{
					temp3[j *subCols + x] = 0;
				}
			}
		}
	
		hipMemcpyAsync(b, temp3, sizeof(float)*subCols*k, hipMemcpyHostToDevice, streams[0]);
		unsigned long long y = 0;
		while(y < numSubMatrixA){
			if(overflowA == 0 && y == numSubMatrixA){
				break;
			}
			for(int j = 0; j < subRows; ++j){
				for(int x = 0; x < k; ++x){
					if(y * subRows + j < m){
						(a_h[y % numStreams])[j * k + x] = A[y*subRows*k + j*k + x];
					}else{
						(a_h[y % numStreams])[j * k + x] = 0;
					}
				}			
			}
			hipMemcpyAsync(a[y % numStreams], a_h[y % numStreams], sizeof(float)*subRows*k, hipMemcpyHostToDevice, streams[y % numStreams]);
			printf("sending multiply %d,%d to stream %d\n", y, i, y % numStreams);
			doMultiply2Matrices(subRows, k, a[y % numStreams], k, subCols, b, c[y % numStreams], streams[y % numStreams], handles[y % numStreams]); 	
			hipMemcpyAsync(c_h[y % numStreams], c[y % numStreams], sizeof(float)*subRows*subCols, hipMemcpyDeviceToHost, streams[y % numStreams]);
			if(y % numStreams == numStreams - 1){
				hipDeviceSynchronize();
				for(int s = 0; s < numStreams; ++s){
					//TODO:Currently does not work with overflowA != 0 or overflowB != 0
					copyElements(C, c_h[s], subRows, subCols, m, n, count * numStreams + s, i);
				}
				++count;
			}
			++y;

		}
		
	
	}

	for(int i = 0; i < numStreams; ++i){
		hipFree(a[i]);
		hipFree(c[i]);
		hipHostFree(a_h[i]);
		hipHostFree(c_h[i]);
		hipStreamDestroy(streams[i]);
	}
	hipFree(b);
	hipHostFree(temp3);
    
}





